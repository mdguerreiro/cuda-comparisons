#define N (512*512)
#define M (10000)

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cudakernel(float *buf)
{
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   buf[i] = 1.0f * i / N;
   for(int j = 0; j < M; j++)
      buf[i] = buf[i] * buf[i] - 0.25f;
}

int main()
{
   float data[N];
   float *d_data;
   hipMalloc(&d_data, N * sizeof(float));
   cudakernel<<<N/256, 256>>>(d_data);
   hipMemcpy(data, d_data, N * sizeof(float), hipMemcpyDeviceToHost);
   hipFree(d_data); 

}
